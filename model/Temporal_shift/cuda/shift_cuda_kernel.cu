#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <math.h>
#include <iostream>

namespace {
  template <typename scalar_t>
  __global__ void shift_cuda_forward_kernel(
      const scalar_t* __restrict__ input,
      scalar_t* output,
      scalar_t* xpos, 
      scalar_t* ypos,
      const int batch,
      const int channel,
      const int bottom_height,
      const int bottom_width,
      const int top_height,
      const int top_width,
      const int stride) 
  {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;


    if (index < batch*channel*top_height*top_width)
    {
	    const int top_sp_dim = top_height * top_width;
	    const int bottom_sp_dim = bottom_height * bottom_width;
	    const int n = index/(channel * top_sp_dim);       
	    const int idx = index%(channel * top_sp_dim);     
	    const int c_out = idx/top_sp_dim;                     
	    const int c_in = c_out;                              
	    const int sp_idx = idx%top_sp_dim;                 
	    const int h = sp_idx/top_width;               
	    const int w = sp_idx%top_width;            
	    const scalar_t* data_im_ptr = input + n*channel*bottom_sp_dim + c_in*bottom_sp_dim; // ->(n,c) 

	    const int h_offset = h * stride;             // h on input feature map
	    const int w_offset = w;              // w on input feature map

	    scalar_t val = 0;
	    const scalar_t x = xpos[c_in];
	    const scalar_t y = ypos[c_in];

	    int h_im, w_im;
	    int x1 = floorf(x);
	    int x2 = x1+1;
	    int y1 = floorf(y);
	    int y2 = y1+1;

	    h_im = h_offset + y1;
	    w_im = w_offset + x1;
	    scalar_t q11 = (h_im >= 0 && w_im >= 0 && h_im < bottom_height && w_im < bottom_width) ? data_im_ptr[h_im*bottom_width + w_im] : 0;

	    h_im = h_offset + y1;
	    w_im = w_offset + x2;
	    scalar_t q21 = (h_im >= 0 && w_im >= 0 && h_im < bottom_height && w_im < bottom_width) ? data_im_ptr[h_im*bottom_width + w_im] : 0;

	    h_im = h_offset + y2;
	    w_im = w_offset + x1;
	    scalar_t q12 = (h_im >= 0 && w_im >= 0 && h_im < bottom_height && w_im < bottom_width) ? data_im_ptr[h_im*bottom_width + w_im] : 0;

	    h_im = h_offset + y2;
	    w_im = w_offset + x2;
	    scalar_t q22 = (h_im >= 0 && w_im >= 0 && h_im < bottom_height && w_im < bottom_width) ? data_im_ptr[h_im*bottom_width + w_im] : 0;

	    scalar_t dx = x-x1;
	    scalar_t dy = y-y1;

	    val = q11*(1-dx)*(1-dy) + q21*dx*(1-dy) + q12*(1-dx)*dy + q22*dx*dy;
	    output[index] = val;
	}
  }

  template <typename scalar_t>
  __global__ void Shift_Bottom_Backward_Stride1(
        const scalar_t* __restrict__ grad_output,
        scalar_t* grad_input,
        scalar_t* xpos,
        scalar_t* ypos,
        const int batch,
        const int channel,
        const int bottom_height,
        const int bottom_width)  
  {
      const int index = blockIdx.x * blockDim.x + threadIdx.x;

	  if (index < batch*channel*bottom_height*bottom_width)
	  {
	      const int top_sp_dim = bottom_height * bottom_width;                // h*w
	      const int bottom_sp_dim = bottom_height * bottom_width;   
	      const int n = index/(channel * bottom_sp_dim);    
	      const int idx = index%(channel * bottom_sp_dim);
	      const int c_in = idx/bottom_sp_dim;
	      const int c_out = c_in;
	      const int sp_idx = idx%bottom_sp_dim;
	      const int h_col = sp_idx/bottom_width;
	      const int w_col = sp_idx%bottom_width;
	      const scalar_t* top_diff_ptr = grad_output + n*channel*top_sp_dim + c_out*top_sp_dim;

	      const int h_offset = h_col;
	      const int w_offset = w_col;

	      scalar_t val = 0;
	      const scalar_t x = -xpos[c_in];  //reverse position
	      const scalar_t y = -ypos[c_in];

	      int h_im, w_im;

	      int x1 = floorf(x);
	      int x2 = x1+1;
	      int y1 = floorf(y);
	      int y2 = y1+1;

	      //q11
	      scalar_t q11 = 0;

	      h_im = (h_offset + y1);
	      w_im = (w_offset + x1);
	      q11 = (h_im >= 0 && w_im >= 0 && h_im < bottom_height && w_im < bottom_width) ? top_diff_ptr[h_im*bottom_width + w_im] : 0;

	      //q21
	      scalar_t q21 = 0;

	      h_im = (h_offset + y1);
	      w_im = (w_offset + x2);
	      q21 = (h_im >= 0 && w_im >= 0 && h_im < bottom_height && w_im < bottom_width) ? top_diff_ptr[h_im*bottom_width + w_im] : 0;

	      //q12
	      scalar_t q12 = 0;

	      h_im = (h_offset + y2);
	      w_im = (w_offset + x1);
	      q12 = (h_im >= 0 && w_im >= 0 && h_im < bottom_height && w_im < bottom_width) ? top_diff_ptr[h_im*bottom_width + w_im] : 0;

	      //q22
	      scalar_t q22 = 0;

	      h_im = (h_offset + y2);
	      w_im = (w_offset + x2);
	      q22 = (h_im >= 0 && w_im >= 0 && h_im < bottom_height && w_im < bottom_width) ? top_diff_ptr[h_im*bottom_width + w_im] : 0;

	      scalar_t dx = x-x1;
	      scalar_t dy = y-y1;

	      val = q11*(1-dx)*(1-dy) + q21*dx*(1-dy) + q12*(1-dx)*dy + q22*dx*dy;
	      grad_input[index] = val;
	}
  } 


  template <typename scalar_t>
  __global__ void Shift_Bottom_Backward(
        const scalar_t* __restrict__ grad_output,
        scalar_t* grad_input,
        scalar_t* xpos,
        scalar_t* ypos,
        const int batch,
        const int channel,
        const int bottom_height,
        const int bottom_width)  
  {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;


    if (index < batch*channel*bottom_height*bottom_width)
    {

	    const int top_height = bottom_height/2;
	    const int top_width = bottom_width;
	    const int stride = 2;
	    const int top_sp_dim = top_height * top_width;
	    const int bottom_sp_dim = bottom_height * bottom_width;
	    const int n = index/(channel * bottom_sp_dim);
	    const int idx = index%(channel * bottom_sp_dim);
	    const int c_in = idx/bottom_sp_dim;
	    const int c_out = c_in;
	    const int sp_idx = idx%bottom_sp_dim;
	    const int h_col = sp_idx/bottom_width;
	    const int w_col = sp_idx%bottom_width;
	    const scalar_t* top_diff_ptr = grad_output + n*channel*top_sp_dim + c_out*top_sp_dim;

	    const int h_offset = h_col;
	    const int w_offset = w_col;


	    scalar_t val = 0;
	    const scalar_t x = -xpos[c_in]; 
	    const scalar_t y = -ypos[c_in];

	    int h_im, w_im;
	    int x1 = floorf(x);
	    int x2 = x1+1;
	    int y1 = floorf(y);
	    int y2 = y1+1;

	    //q11
	    scalar_t q11 = 0;

	    h_im = (h_offset + y1);
	    w_im = (w_offset + x1);
	    if(h_im%stride == 0)
	    {
	      h_im=h_im/stride;

	      q11 = (h_im >= 0 && w_im >= 0 && h_im < top_height && w_im < top_width) ? top_diff_ptr[h_im*top_width + w_im] : 0;
	    }

	    //q21
	    scalar_t q21 = 0;

	    h_im = (h_offset + y1);
	    w_im = (w_offset + x2);
	    if(h_im%stride == 0)
	    {
	      h_im=h_im/stride;

	      q21 = (h_im >= 0 && w_im >= 0 && h_im < top_height && w_im < top_width) ? top_diff_ptr[h_im*top_width + w_im] : 0;
	    }

	    //q12
	    scalar_t q12 = 0;

	    h_im = (h_offset + y2);
	    w_im = (w_offset + x1);

	    if(h_im%stride == 0)
	    {
	      h_im=h_im/stride;

	      q12 = (h_im >= 0 && w_im >= 0 && h_im < top_height && w_im < top_width) ? top_diff_ptr[h_im*top_width + w_im] : 0;
	    }

	    //q22
	    scalar_t q22 = 0;

	    h_im = (h_offset + y2);
	    w_im = (w_offset + x2);

	    if(h_im%stride == 0)
	    {
	      h_im=h_im/stride;

	      q22 = (h_im >= 0 && w_im >= 0 && h_im < top_height && w_im < top_width) ? top_diff_ptr[h_im*top_width + w_im] : 0;
	    }

	    scalar_t dx = x-x1;
	    scalar_t dy = y-y1;

	    val = q11*(1-dx)*(1-dy) + q21*dx*(1-dy) + q12*(1-dx)*dy + q22*dx*dy;
	    grad_input[index] = val;
	}
  } // namespace



  template <typename scalar_t>
  __inline__ __device__ void myAtomicAdd(scalar_t *buf, scalar_t val);

  template <>
  __inline__ __device__ void myAtomicAdd<float>(float *buf, float val)
  {
    atomicAdd(buf, val);
  }

  template <>
  __inline__ __device__ void myAtomicAdd<double>(double *buf, double val)
  {
    //Not Supported
  }



  template <typename scalar_t>
  __global__ void Shift_Position_Backward(
        const scalar_t* __restrict__ input,
        const scalar_t* __restrict__ grad_output,
        scalar_t* grad_input,
        scalar_t* xpos,
        scalar_t* ypos,
        scalar_t* grad_xpos_bchw,
        scalar_t* grad_ypos_bchw,
        const int batch,
        const int channel,
        const int bottom_height,
        const int bottom_width,
        const int stride)  
  {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    const int top_height = bottom_height/stride;
    const int top_width = bottom_width;


    if (index < batch*channel*top_height*top_width)
    {
	    const int top_sp_dim = top_height * top_width;
	    const int bottom_sp_dim = bottom_height * bottom_width;
	    const int n = index/(channel * top_sp_dim);
	    const int idx = index%(channel * top_sp_dim);
	    const int c_mul = 1;
	    const int c_out = idx/top_sp_dim;
	    const int c_in = c_out/c_mul;
	    const int sp_idx = idx%top_sp_dim;
	    const int h = sp_idx/top_width;
	    const int w = sp_idx%top_width;
	    const scalar_t* data_im_ptr = input + n*channel*bottom_sp_dim + c_in*bottom_sp_dim;

	    const int h_offset = h * stride;
	    const int w_offset = w;

	    //output : 2*(C) x (1*H*W)
	    const int kernel_offset = top_sp_dim;
	    const int c_off = c_out % c_mul;

	    scalar_t val_x = 0, val_y = 0;

	    const scalar_t shiftX = xpos[c_in];
	    const scalar_t shiftY = ypos[c_in];


	    const int ix1 = floorf(shiftX);
	    const int ix2 = ix1+1;
	    const int iy1 = floorf(shiftY);
	    const int iy2 = iy1+1;
	    const scalar_t dx = shiftX-ix1;
	    const scalar_t dy = shiftY-iy1;

	    const int h_im1 = h_offset + iy1;
	    const int h_im2 = h_offset + iy2;

	    const int w_im1 = w_offset + ix1;
	    const int w_im2 = w_offset + ix2;

	    const scalar_t q11 = (h_im1 >= 0 && w_im1 >= 0 && h_im1 < bottom_height && w_im1 < bottom_width) ? data_im_ptr[h_im1*bottom_width + w_im1] : 0;
	    const scalar_t q21 = (h_im1 >= 0 && w_im2 >= 0 && h_im1 < bottom_height && w_im2 < bottom_width) ? data_im_ptr[h_im1*bottom_width + w_im2] : 0;
	    const scalar_t q12 = (h_im2 >= 0 && w_im1 >= 0 && h_im2 < bottom_height && w_im1 < bottom_width) ? data_im_ptr[h_im2*bottom_width + w_im1] : 0;
	    const scalar_t q22 = (h_im2 >= 0 && w_im2 >= 0 && h_im2 < bottom_height && w_im2 < bottom_width) ? data_im_ptr[h_im2*bottom_width + w_im2] : 0;

	    val_x = (1-dy)*(q21-q11)+dy*(q22-q12);
	    val_y = (1-dx)*(q12-q11)+dx*(q22-q21);



	  	grad_xpos_bchw[index] = val_x * grad_output[index];
	    grad_ypos_bchw[index] = val_y * grad_output[index];

	  	//grad_xpos_bchw[index] = val_x;
	    //grad_ypos_bchw[index] = val_y;

	    //grad_xpos_bchw[index] = 0;
	    //grad_ypos_bchw[index] = 0;

	    //scalar_t* out_ptr_x = grad_xpos_bchw + index;
	    //scalar_t* out_ptr_y = grad_ypos_bchw + index;

	    //myAtomicAdd(out_ptr_x, val_x * grad_output[index]);
	    //myAtomicAdd(out_ptr_y, val_y * grad_output[index]);
	}
  } // namespace






  template <typename scalar_t>
  __global__ void applyShiftConstraint(
        scalar_t* grad_xpos,
        scalar_t* grad_ypos,
        const int channel)  
  {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < channel)
    {
	  const scalar_t dx = grad_xpos[index];
	  const scalar_t dy = grad_ypos[index];
	  const scalar_t dr = sqrt(dy*dy);

	  if(dr!=0)
	  {
		  grad_xpos[index] = dx/dr*0.0;
		  grad_ypos[index] = dy/dr*0.01;
	  }
	  else                                  // without this, the grad_ypos may be large.
	  {
		  grad_xpos[index] = 0.0;
		  grad_ypos[index] = 0.0001;
	  }
	}
  } // namespace




}




at::Tensor shift_cuda_forward(
    at::Tensor input,at::Tensor xpos,at::Tensor ypos,const int stride) {

  auto output = at::zeros({input.size(0), input.size(1), input.size(2)/stride, input.size(3)}, input.options());

  const dim3 blocks((input.size(0)*input.size(1)*input.size(2)*input.size(3)/stride+1024-1)/1024);
  const int threads = 1024;

  AT_DISPATCH_FLOATING_TYPES(input.type(), "shift_forward_cuda", ([&] {
    shift_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
      input.data<scalar_t>(),
      output.data<scalar_t>(),
      xpos.data<scalar_t>(),
      ypos.data<scalar_t>(),
      input.size(0),
      input.size(1),
      input.size(2),
      input.size(3),
      input.size(2)/stride,
      input.size(3),
      stride);
  }));

  //std::cout << output[0] << std::endl;

  return output;
}

std::vector<at::Tensor> shift_cuda_backward(
    at::Tensor grad_output,
    at::Tensor input,
    at::Tensor output,
    at::Tensor xpos,
    at::Tensor ypos,
    const int stride) {
  auto grad_input = at::zeros_like(input);




  const dim3 blocks((input.size(0)*input.size(1)*input.size(2)*input.size(3)+1024-1)/1024);
  const int threads = 1024;

  if(stride==1)
  {
    AT_DISPATCH_FLOATING_TYPES(input.type(), "Shift_Bottom_Backward_Stride1_", ([&] {
      Shift_Bottom_Backward_Stride1<scalar_t><<<blocks, threads>>>(
        grad_output.data<scalar_t>(),
        grad_input.data<scalar_t>(),
        xpos.data<scalar_t>(),
        ypos.data<scalar_t>(),
        input.size(0),
        input.size(1),
        input.size(2),
        input.size(3));
    }));
  }
  else
  {
    AT_DISPATCH_FLOATING_TYPES(input.type(), "Shift_Bottom_Backward_", ([&] {
      Shift_Bottom_Backward<scalar_t><<<blocks, threads>>>(
        grad_output.data<scalar_t>(),
        grad_input.data<scalar_t>(),
        xpos.data<scalar_t>(),
        ypos.data<scalar_t>(),
        input.size(0),
        input.size(1),
        input.size(2),
        input.size(3));
    }));
  }




  auto grad_xpos_bchw = at::zeros({output.size(0), output.size(1), output.size(2), output.size(3)}, output.options()); // (b,c,h,w)
  auto grad_ypos_bchw = at::zeros({output.size(0), output.size(1), output.size(2), output.size(3)}, output.options()); // (b,c,h,w)

  const dim3 blocks_output((output.size(0)*output.size(1)*output.size(2)*output.size(3)+1024-1)/1024);

  AT_DISPATCH_FLOATING_TYPES(input.type(), "Shift_Position_Backward_", ([&] {
    Shift_Position_Backward<scalar_t><<<blocks_output, threads>>>(
      input.data<scalar_t>(),
      grad_output.data<scalar_t>(),
      grad_input.data<scalar_t>(),
      xpos.data<scalar_t>(),
      ypos.data<scalar_t>(),
      grad_xpos_bchw.data<scalar_t>(),
      grad_ypos_bchw.data<scalar_t>(),
      input.size(0),
      input.size(1),
      input.size(2),
      input.size(3),
      stride);
  }));

  auto grad_xpos_chw = at::mean(grad_xpos_bchw, 0, false);
  auto grad_xpos_ch = at::sum(grad_xpos_chw, 2, false);
  auto grad_xpos_c  = at::sum(grad_xpos_ch, 1, false);
  auto grad_xpos = grad_xpos_c;

  auto grad_ypos_chw = at::mean(grad_ypos_bchw, 0, false);
  auto grad_ypos_ch = at::sum(grad_ypos_chw, 2, false);
  auto grad_ypos_c  = at::sum(grad_ypos_ch, 1, false);
  auto grad_ypos = grad_ypos_c;
  


  const dim3 blocks_norm((output.size(1)+1024-1)/1024);

  AT_DISPATCH_FLOATING_TYPES(input.type(), "applyShiftConstraint_", ([&] {
    applyShiftConstraint<scalar_t><<<blocks_norm, threads>>>(
      grad_xpos.data<scalar_t>(),
      grad_ypos.data<scalar_t>(),
      output.size(1));
  }));

  return {grad_input,grad_xpos,grad_ypos};
}
